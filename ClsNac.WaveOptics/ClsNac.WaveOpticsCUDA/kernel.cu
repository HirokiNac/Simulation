#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>


const int threads = 16;
const int numBlock = 16;
const dim3 threadsPerBlock = dim3(512, 512);

dim3 calcBlock(dim3 thread, int x, int y)
{
	return dim3(
		static_cast<int>(ceil(1.0*x / thread.x)),
		static_cast<int>(ceil(1.0*y / thread.y)));
}


__global__ void PropFw1D(const double _k,
	const int _n1, const double* _x1, const double* _y1, const double* _u1re, const double* _u1im,
	const int _n2, const double* _x2, const double* _y2, double* _u2re, double* _u2im)
{
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	double r, rx, ry, rr;
	double tr, ti;
	double tur, tui;
	double ur = 0.0, ui = 0.0;
	if (col < _n2)
	{
		for (int j = 0; j < _n1; j++)
		{
			rx = _x2[col] - _x1[j];
			ry = _y2[col] - _y1[j];
			r = sqrt(rx*rx + ry*ry);

			rr = 1.0 / sqrt(r);
			tr = cos(-_k*r) * rr;
			ti = sin(-_k*r) * rr;

			tur = _u1re[j];
			tui = _u1im[j];

			ur = ur + tur*tr - tui*ti;
			ui = ui + tur*ti + tui*tr;

		}
		_u2re[col] = _u2re[col] + ur;
		_u2im[col] = _u2im[col] + ui;

	}

}

__global__ void PropFw1D_f(const float _k,
	const int _n1, const float* _x1, const float* _y1, const float* _u1re, const float* _u1im,
	const int _n2, const float* _x2, const  float* _y2, float* _u2re, float* _u2im)
{
	const unsigned int col = blockIdx.x + blockDim.x* threadIdx.x;

	float r, rx, ry;
	float tr, ti;
	float tur, tui;
	float ur, ui;
	for (int j = 0; j < _n1; j++)
	{
		rx = _x2[col] - _x1[j];
		ry = _y2[col] - _y1[j];
		r = sqrt(rx*rx + ry*ry);

		tr = cos(-_k*r) / r;
		ti = sin(-_k*r) / r;

		tur = _u1re[j];
		tui = _u1im[j];

		tur = ur + tur*tr - tui*ti;
		tui = ui + tur*ti + tui*tr;

	}
	_u2re[col] = _u2re[col] + ur;
	_u2im[col] = _u2im[col] + ui;

}

__global__ void PropFw2D(const double _k,
	const int _n1, const double* _x1, const double* _y1, const  double* _z1, const  double* _u1re, const  double* _u1im,
	const int _n2, const double* _x2, const double* _y2, const double* _z2, double* _u2re, double* _u2im)
{
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	//
	if (_n2 < col)return;

	//
	double r, rx, ry, rz, rr;
	double tr, ti;
	double tur, tui;
	double ur = 0.0, ui = 0.0;
	double x1, y1, z1;

	for (int j = 0; j < _n1; j++)
	{
		x1 = _x1[j];
		y1 = _y1[j];
		z1 = _z1[j];
		
		rx = _x2[col] - x1;
		ry = _y2[col] - y1;
		rz = _z2[col] - z1;
		r = sqrt(rx*rx + ry*ry + rz*rz);

		rr = 1.0 / r;
		tr = cos(-_k*r) * rr;
		ti = sin(-_k*r) * rr;
		
		tur = _u1re[j];
		tui = _u1im[j];

		ur = ur + tur*tr - tui*ti;
		ui = ui + tur*ti + tui*tr;

	}
	_u2re[col] = _u2re[col] + ur;
	_u2im[col] = _u2im[col] + ui;
	__syncthreads();
}

__global__ void PropFw2D2(const double _k,
	const int _m1,const int _n1, const double* _x1, const double* _y1, const  double* _z1, const  double* _u1re, const  double* _u1im,
	const int _m2,const int _n2, const double* _x2, const double* _y2, const double* _z2, double* _u2re, double* _u2im)
{
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
	//
	if (_m2 < col || _n2 < row)return;
	const unsigned int colrow = col*_n2 + row;
	//
	double r, rx, ry, rz, rr;
	double tr, ti;
	double tur, tui;
	double ur = 0.0, ui = 0.0;
	double x1, y1, z1;

	for (int j = 0; j < _n1; j++)
	{
		x1 = _x1[j];
		y1 = _y1[j];
		z1 = _z1[j];

		rx = _x2[colrow] - x1;
		ry = _y2[colrow] - y1;
		rz = _z2[colrow] - z1;
		r = sqrt(rx*rx + ry*ry + rz*rz);

		rr = 1.0 / r;
		tr = cos(-_k*r) * rr;
		ti = sin(-_k*r) * rr;

		tur = _u1re[j];
		tui = _u1im[j];

		ur = ur + tur*tr - tui*ti;
		ui = ui + tur*ti + tui*tr;

	}
	_u2re[colrow] = _u2re[colrow] + ur;
	_u2im[colrow] = _u2im[colrow] + ui;
	__syncthreads();
}

__global__ void PropFw2D_f(const float _k,
	const int _n1, const float* _x1, const float* _y1, const  float* _z1, const  float* _u1re, const  float* _u1im,
	const int _n2, const float* _x2, const float* _y2, const float* _z2, float* _u2re, float* _u2im)
{
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	//
	if (_n2 < col)return;

	//
	float r, rx, ry, rz, rr;
	float tr, ti;
	float tur, tui;
	float ur = 0.0, ui = 0.0;
	float x1, y1, z1;

	for (int j = 0; j < _n1; j++)
	{
		x1 = _x1[j];
		y1 = _y1[j];
		z1 = _z1[j];

		rx = _x2[col] - x1;
		ry = _y2[col] - y1;
		rz = _z2[col] - z1;
		r = sqrt(rx*rx + ry*ry + rz*rz);

		rr = 1.0 / r;
		tr = cos(-_k*r) * rr;
		ti = sin(-_k*r) * rr;

		tur = _u1re[j];
		tui = _u1im[j];

		ur = ur + tur*tr - tui*ti;
		ui = ui + tur*ti + tui*tr;

	}
	_u2re[col] = _u2re[col] + ur;
	_u2im[col] = _u2im[col] + ui;
	__syncthreads();
}

extern "C" void
PropFw1dCuda(double _k,
int _n1, double* _x1, double* _y1, double* _u1re, double* _u1im,
int _n2, double* _x2, double* _y2, double* &_u2re, double* &_u2im)
{
	hipSetDevice(0);

	size_t memsize1 = _n1*sizeof(double);
	size_t memsize2 = _n2*sizeof(double);

	//1
	double *dx1 = 0;
	hipMalloc((void**)&dx1, memsize1);
	hipMemcpy(dx1, _x1, memsize1, hipMemcpyHostToDevice);

	double *dy1 = 0;
	hipMalloc((void**)&dy1, memsize1);
	hipMemcpy(dy1, _y1, memsize1, hipMemcpyHostToDevice);

	double *du1re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du1re, _u1re, memsize1, hipMemcpyHostToDevice);

	double *du1im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du1im, _u1im, memsize1, hipMemcpyHostToDevice);

	//2
	double *dx2 = 0;
	hipMalloc((void**)&dx2, memsize2);
	hipMemcpy(dx2, _x2, memsize2, hipMemcpyHostToDevice);

	double *dy2 = 0;
	hipMalloc((void**)&dy2, memsize2);
	hipMemcpy(dy2, _y2, memsize2, hipMemcpyHostToDevice);

	double *du2re = 0;
	hipMalloc((void**)&du2re, memsize2);
	//hipMemcpy(du2re, _u2re, memsize2, hipMemcpyHostToDevice);

	double *du2im = 0;
	hipMalloc((void**)&du2im, memsize2);
	//hipMemcpy(du2im, _u2im, memsize2, hipMemcpyHostToDevice);

	PropFw1D << <numBlock, threadsPerBlock >> >(_k, _n1, dx1, dy1, du1re, du1im, _n2, dx2, dy2, du2re, du2im);

	double* u2re_out = 0;
	hipHostMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, du2re, memsize2, hipMemcpyDeviceToHost);
	double* u2im_out = 0;
	hipHostMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, du2im, memsize2, hipMemcpyDeviceToHost);


	for (int i = 0; i < _n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}


	//memfree
	hipFree(dx1);
	hipFree(dy1);
	hipFree(du1re);
	hipFree(du1im);

	hipFree(dx2);
	hipFree(dy2);
	hipFree(du2re);
	hipFree(du2im);
	hipFree(u2re_out);
	hipFree(u2im_out);
}

extern "C" void
PropFw1dCuda_f(float _k,
int _n1, float* _x1,float* _y1, float* _u1re, float* _u1im,
int _n2, float* _x2, float* _y2, float* &_u2re, float* &_u2im)
{
	hipSetDevice(0);

	size_t memsize1 = _n1*sizeof(float);
	size_t memsize2 = _n2*sizeof(float);

	//1
	float *dx1 = 0;
	hipMalloc((void**)&dx1, memsize1);
	hipMemcpy(dx1, _x1, memsize1, hipMemcpyHostToDevice);

	float *dy1 = 0;
	hipMalloc((void**)&dy1, memsize1);
	hipMemcpy(dy1, _y1, memsize1, hipMemcpyHostToDevice);

	float *du1re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du1re, _u1re, memsize1, hipMemcpyHostToDevice);

	float *du1im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du1im, _u1im, memsize1, hipMemcpyHostToDevice);

	//2
	float *dx2 = 0;
	hipMalloc((void**)&dx2, memsize2);
	hipMemcpy(dx2, _x2, memsize2, hipMemcpyHostToDevice);

	float *dy2 = 0;
	hipMalloc((void**)&dy2, memsize2);
	hipMemcpy(dy2, _y2, memsize2, hipMemcpyHostToDevice);

	float *du2re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du2re, _u2re, memsize1, hipMemcpyHostToDevice);

	float *du2im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du2im, _u2im, memsize1, hipMemcpyHostToDevice);

	PropFw1D_f << <_n2 / 512, 512 >> >(_k, _n1, dx1, dy1, du1re, du1im, _n2, dx2, dy2, du2re, du2im);

	//out
	float* u2re_out = (float*)malloc(memsize2);
	//hipHostMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, du2re, memsize2, hipMemcpyDeviceToHost);

	float* u2im_out = (float*)malloc(memsize2);
	//hipHostMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, du2im, memsize2, hipMemcpyDeviceToHost);

	for (int i = 0; i < _n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}

	//memfree
	//1
	hipFree(dx1);
	hipFree(dy1);
	hipFree(du1re);
	hipFree(du1im);
	//2
	hipFree(dx2);
	hipFree(dy2);
	hipFree(du2re);
	hipFree(du2im);
	//out
	free(u2re_out);
	free(u2im_out);
	//hipFree(u2re_out);
	//hipFree(u2im_out);
}

extern "C" void
PropFw2dCuda(double _k,
int _n1, double* _x1, double* _y1, double* _z1, double* _u1re, double* _u1im,
int _n2, double* _x2, double* _y2, double* _z2, double* &_u2re, double* &_u2im)
{
	hipSetDevice(1);

	size_t memsize1 = _n1*sizeof(double);
	size_t memsize2 = _n2*sizeof(double);

	//1
	double *dx1 = 0;
	hipMalloc((void**)&dx1, memsize1);
	hipMemcpy(dx1, _x1, memsize1, hipMemcpyHostToDevice);

	double *dy1 = 0;
	hipMalloc((void**)&dy1, memsize1);
	hipMemcpy(dy1, _y1, memsize1, hipMemcpyHostToDevice);

	double *dz1 = 0;
	hipMalloc((void**)&dz1, memsize1);
	hipMemcpy(dz1, _z1, memsize1, hipMemcpyHostToDevice);

	double *du1re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du1re, _u1re, memsize1, hipMemcpyHostToDevice);

	double *du1im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du1im, _u1im, memsize1, hipMemcpyHostToDevice);


	//2
	double *dx2 = 0;
	hipMalloc((void**)&dx2, memsize2);
	hipMemcpy(dx2, _x2, memsize2, hipMemcpyHostToDevice);

	double *dy2 = 0;
	hipMalloc((void**)&dy2, memsize2);
	hipMemcpy(dy2, _y2, memsize2, hipMemcpyHostToDevice);

	double *dz2 = 0;
	hipMalloc((void**)&dz2, memsize2);
	hipMemcpy(dz2, _z2, memsize2, hipMemcpyHostToDevice);

	double *du2re = 0;
	hipMalloc((void**)&du2re, memsize2);
	//hipMemcpy(du2re, _u2re, memsize2, hipMemcpyHostToDevice);

	double *du2im = 0;
	hipMalloc((void**)&du2im, memsize2);
	//hipMemcpy(du2im, _u2im, memsize2, hipMemcpyHostToDevice);
	dim3 b = calcBlock(threadsPerBlock, _n2, 1);
	PropFw2D << <_n2/512,512 /*calcBlock(threadsPerBlock,_n2,1), threadsPerBlock*/ >> >(_k, _n1, dx1, dy1, dz1, du1re, du1im, _n2, dx2, dy2, dz2, du2re, du2im);

	hipDeviceSynchronize();


	double* u2re_out =  (double*)malloc(memsize2);
	//hipMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, du2re, memsize2, hipMemcpyDeviceToHost);
	double* u2im_out =   (double*)malloc(memsize2);
	//hipMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, du2im, memsize2, hipMemcpyDeviceToHost);


	for (int i = 0; i < _n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}

	//memfree
	hipFree(dx1);
	hipFree(dy1);
	hipFree(dz1);
	hipFree(du1re);
	hipFree(du1im);

	hipFree(dx2);
	hipFree(dy2);
	hipFree(dz2);
	hipFree(du2re);
	hipFree(du2im);

	free(u2re_out);
	free(u2im_out);
	//hipHostFree(u2re_out);
	//hipHostFree(u2im_out);

	hipDeviceReset();
}

extern "C" void
PropFw2dCuda2(double _k,
int _m1,int _n1, double* _x1, double* _y1, double* _z1, double* _u1re, double* _u1im,
int _m2,int _n2, double* _x2, double* _y2, double* _z2, double* &_u2re, double* &_u2im)
{
	hipSetDevice(1);

	size_t memsize1 = _m1*_n1*sizeof(double);
	size_t memsize2 = _m2*_n2*sizeof(double);

	//1
	double *dx1 = 0;
	hipMalloc((void**)&dx1, memsize1);
	hipMemcpy(dx1, _x1, memsize1, hipMemcpyHostToDevice);

	double *dy1 = 0;
	hipMalloc((void**)&dy1, memsize1);
	hipMemcpy(dy1, _y1, memsize1, hipMemcpyHostToDevice);

	double *dz1 = 0;
	hipMalloc((void**)&dz1, memsize1);
	hipMemcpy(dz1, _z1, memsize1, hipMemcpyHostToDevice);

	double *du1re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du1re, _u1re, memsize1, hipMemcpyHostToDevice);

	double *du1im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du1im, _u1im, memsize1, hipMemcpyHostToDevice);


	//2
	double *dx2 = 0;
	hipMalloc((void**)&dx2, memsize2);
	hipMemcpy(dx2, _x2, memsize2, hipMemcpyHostToDevice);

	double *dy2 = 0;
	hipMalloc((void**)&dy2, memsize2);
	hipMemcpy(dy2, _y2, memsize2, hipMemcpyHostToDevice);

	double *dz2 = 0;
	hipMalloc((void**)&dz2, memsize2);
	hipMemcpy(dz2, _z2, memsize2, hipMemcpyHostToDevice);

	double *du2re = 0;
	hipMalloc((void**)&du2re, memsize2);
	//hipMemcpy(du2re, _u2re, memsize2, hipMemcpyHostToDevice);

	double *du2im = 0;
	hipMalloc((void**)&du2im, memsize2);
	//hipMemcpy(du2im, _u2im, memsize2, hipMemcpyHostToDevice);

	PropFw2D2 << <calcBlock(threadsPerBlock, _m2, _n2), threadsPerBlock >> >(_k, _m1, _n1, dx1, dy1, dz1, du1re, du1im, _m2, _n2, dx2, dy2, dz2, du2re, du2im);

	hipDeviceSynchronize();


	double* u2re_out = (double*)malloc(memsize2);
	//hipMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, du2re, memsize2, hipMemcpyDeviceToHost);
	double* u2im_out = (double*)malloc(memsize2);
	//hipMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, du2im, memsize2, hipMemcpyDeviceToHost);


	for (int i = 0; i < _m2*_n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}

	//memfree
	hipFree(dx1);
	hipFree(dy1);
	hipFree(dz1);
	hipFree(du1re);
	hipFree(du1im);

	hipFree(dx2);
	hipFree(dy2);
	hipFree(dz2);
	hipFree(du2re);
	hipFree(du2im);

	free(u2re_out);
	free(u2im_out);
	//hipHostFree(u2re_out);
	//hipHostFree(u2im_out);

	hipDeviceReset();
}

extern "C" void
PropFw2dCuda_f(float _k,
int _n1, float* _x1, float* _y1, float* _z1, float* _u1re, float* _u1im,
int _n2, float* _x2, float* _y2, float* _z2, float* &_u2re, float* &_u2im)
{
	hipSetDevice(1);

	size_t memsize1 = _n1*sizeof(float);
	size_t memsize2 = _n2*sizeof(float);

	//1
	float *dx1 = 0;
	hipMalloc((void**)&dx1, memsize1);
	hipMemcpy(dx1, _x1, memsize1, hipMemcpyHostToDevice);

	float *dy1 = 0;
	hipMalloc((void**)&dy1, memsize1);
	hipMemcpy(dy1, _y1, memsize1, hipMemcpyHostToDevice);

	float *dz1 = 0;
	hipMalloc((void**)&dz1, memsize1);
	hipMemcpy(dz1, _z1, memsize1, hipMemcpyHostToDevice);

	float *du1re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du1re, _u1re, memsize1, hipMemcpyHostToDevice);

	float *du1im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du1im, _u1im, memsize1, hipMemcpyHostToDevice);


	//2
	float *dx2 = 0;
	hipMalloc((void**)&dx2, memsize2);
	hipMemcpy(dx2, _x2, memsize2, hipMemcpyHostToDevice);

	float *dy2 = 0;
	hipMalloc((void**)&dy2, memsize2);
	hipMemcpy(dy2, _y2, memsize2, hipMemcpyHostToDevice);

	float *dz2 = 0;
	hipMalloc((void**)&dz2, memsize2);
	hipMemcpy(dz2, _z2, memsize2, hipMemcpyHostToDevice);

	float *du2re = 0;
	hipMalloc((void**)&du2re, memsize2);
	//hipMemcpy(du2re, _u2re, memsize2, hipMemcpyHostToDevice);

	float *du2im = 0;
	hipMalloc((void**)&du2im, memsize2);
	//hipMemcpy(du2im, _u2im, memsize2, hipMemcpyHostToDevice);

	PropFw2D_f << <_n2 / 512, 512>> >(_k, _n1, dx1, dy1, dz1, du1re, du1im, _n2, dx2, dy2, dz2, du2re, du2im);

	hipDeviceSynchronize();


	float* u2re_out = (float*)malloc(memsize2);
	//hipHostMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, du2re, memsize2, hipMemcpyDeviceToHost);
	float* u2im_out =   (float*)malloc(memsize2);
	//hipHostMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, du2im, memsize2, hipMemcpyDeviceToHost);


	for (int i = 0; i < _n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}

	//memfree
	hipFree(dx1);
	hipFree(dy1);
	hipFree(dz1);
	hipFree(du1re);
	hipFree(du1im);

	hipFree(dx2);
	hipFree(dy2);
	hipFree(dz2);
	hipFree(du2re);
	hipFree(du2im);

	free(u2re_out);
	free(u2im_out);
	//hipHostFree(u2re_out);
	//hipHostFree(u2im_out);

	hipDeviceReset();
}
